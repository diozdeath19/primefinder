#include "hip/hip_runtime.h"
#define OUTPUT "output"
#define BLOCK_SIZE 1024

#include <iostream>
#include <string>
#include <fstream>	//Writing to files
#include <chrono>	//Keep track of time
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
// to remove intellisense highlighting
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <algorithm>
#include ""

#include "boinc_api.h"
#include <stdio.h>

using namespace std::chrono;


__global__ static void FindPrimes(char *num, long start, long end){
	long id = blockIdx.x * blockDim.x + threadIdx.x + start;
  if(id >= 2 && id <= end && id != 2 && id != 3 && id != 5 && id != 7) {
    if((id % 2 == 0) || (id % 3 == 0) || (id % 5 == 0) || (id % 7 == 0)) {
      num[id - start] = '1';
    }
  }
}

int main(int argc, char* argv[]) {
	long start_range = 2;
  long end_range = 500000;
  for (int i=0; i<argc; i++) {
    if (!strcmp(argv[i], "-start")) {
      start_range = atol(argv[++i]);
    }
    if (!strcmp(argv[i], "-end")) {
      end_range = atol(argv[++i]);
    }
  }

  if(start_range >= end_range) {
    return 0;
  }

  int retval;
  char buf[256], output_path[512];
  MFILE out;
  double procent = 100;

  retval = boinc_init();
  if (retval) {
      fprintf(stderr,
          "%s boinc_init returned %d\n",
          boinc_msg_prefix(buf, sizeof(buf)), retval
      );
      exit(retval);
  }

  char *gpudata;
  long range = end_range - start_range;
  char *cpudata = new char[range]();

  //memset(cpudata, '0', range);

	//Allocate memory
	hipMalloc((void**)&gpudata, sizeof(char)*range);

	//Copy to GPU
	hipMemcpy(gpudata, cpudata, sizeof(char)*range, hipMemcpyHostToDevice);
	
	float gpu_elapsed_time_ms;
  // some events to count the execution time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  long GRID_SIZE = (range + BLOCK_SIZE - 1) / BLOCK_SIZE;

  hipEventRecord(start, 0);
	//Kernel call on the GPU
	FindPrimes << <GRID_SIZE, BLOCK_SIZE >> >(gpudata, start_range, end_range);
	
	//Synchronize the device and the host
	hipDeviceSynchronize();

	//Copy from GPU back onto host
	hipMemcpy(cpudata, gpudata, sizeof(char)*range, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
  hipEventSynchronize(stop);

  // compute time elapse on GPU computing
  hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

  /*  
  for (long i = start_range; i < end_range; i++) {
    if(cpudata[i - start_range] == '\0') {
      printf("%ld\t", i);
    }
	}
  */
  
	//Free the memory on the GPU
	hipFree(gpudata);

	//Reset the device for easy profiling
	hipDeviceReset();

  boinc_resolve_filename(OUTPUT, output_path, sizeof(output_path));

  retval = out.open(output_path, "a+");
  
  if (retval) {
      fprintf(stderr,
          "%s APP:  output open failed:\n",
          boinc_msg_prefix(buf, sizeof(buf))
      );
      fprintf(stderr,
          "%s resolved name %s, retval %d\n",
          boinc_msg_prefix(buf, sizeof(buf)), output_path, retval
      );
      perror("open");
      exit(1);
  }
  
  out.printf("Time elapsed on prime finding of range from %ld to %ld = %f ms\n\n", start_range, end_range, gpu_elapsed_time_ms);
  retval = out.flush(); //force the output file to be closed.
  if (retval) {
      fprintf(stderr,
          "%s APP: primefinder flush failed %d\n",
          boinc_msg_prefix(buf, sizeof(buf)), retval
      );
      exit(1);
  }

  boinc_fraction_done(procent);

  boinc_finish(0);

	return 0;
}